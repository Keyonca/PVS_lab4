#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void array_ops_parallel(double* a, double* b, double* sum,
                                   double* diff, double* prod, double* div, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        sum[idx] = a[idx] + b[idx];
        diff[idx] = a[idx] - b[idx];
        prod[idx] = a[idx] * b[idx];
        div[idx] = a[idx] / b[idx];
    }
}

int main(int argc, char* argv[]) {
    int N = 200000;      // Значение по умолчанию
    int block_size = 256; // Значение по умолчанию

    // Обработка параметров: приоритет у аргументов командной строки
    if (argc > 1) {
        N = atoi(argv[1]);
    } else {
        char *env_n = getenv("ARRAY_SIZE");
        if (env_n != NULL) {
            N = atoi(env_n);
        }
    }

    if (argc > 2) {
        block_size = atoi(argv[2]);
    } else {
        char *env_bs = getenv("BLOCK_SIZE");
        if (env_bs != NULL) {
            block_size = atoi(env_bs);
        }
    }

    if (N < 100000) {
        fprintf(stderr, "N must be >= 100000\n");
        return 1;
    }

    double *a, *b, *sum, *diff, *prod, *div;
    double *d_a, *d_b, *d_sum, *d_diff, *d_prod, *d_div;

    // Выделение памяти на хосте
    a = (double*)malloc(N * sizeof(double));
    b = (double*)malloc(N * sizeof(double));
    sum = (double*)malloc(N * sizeof(double));
    diff = (double*)malloc(N * sizeof(double));
    prod = (double*)malloc(N * sizeof(double));
    div = (double*)malloc(N * sizeof(double));

    // Инициализация массивов
    for(int i = 0; i < N; i++) {
        a[i] = i + 1.0;
        b[i] = (i + 1.0) * 2.0;
    }

    // Выделение памяти на устройстве
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_sum, N * sizeof(double));
    hipMalloc(&d_diff, N * sizeof(double));
    hipMalloc(&d_prod, N * sizeof(double));
    hipMalloc(&d_div, N * sizeof(double));

    // Копирование данных на устройство
    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    // Расчет конфигурации запуска
    int grid_size = (N + block_size - 1) / block_size;

    // Создание событий для замера времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Прогревочный запуск
    array_ops_parallel<<<grid_size, block_size>>>(d_a, d_b, d_sum, d_diff, d_prod, d_div, N);
    hipDeviceSynchronize();

    // Замер времени выполнения
    hipEventRecord(start);
    array_ops_parallel<<<grid_size, block_size>>>(d_a, d_b, d_sum, d_diff, d_prod, d_div, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Проверка ошибок CUDA
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Расчет времени выполнения
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Вывод результатов
    printf("ВРЕМЯ_ВЫПОЛНЕНИЯ: %.6f s\n", milliseconds / 1000.0f);
    printf("ПАРАЛЛЕЛЬНАЯ_РЕАЛИЗАЦИЯ\n");
    printf("ЭЛЕМЕНТОВ: %d\n", N);
    printf("БЛОКОВ: %d\n", grid_size);
    printf("ПОТОКОВ_В_БЛОКЕ: %d\n", block_size);
    printf("ОБЩЕЕ_КОЛИЧЕСТВО_ПОТОКОВ: %d\n", grid_size * block_size);

    // Освобождение памяти
    free(a); free(b); free(sum); free(diff); free(prod); free(div);
    hipFree(d_a); hipFree(d_b); hipFree(d_sum);
    hipFree(d_diff); hipFree(d_prod); hipFree(d_div);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
