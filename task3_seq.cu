#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void array_ops_sequential(double* a, double* b, double* sum,
                                    double* diff, double* prod, double* div, int n) {
    for(int i = 0; i < n; i++) {
        sum[i] = a[i] + b[i];
        diff[i] = a[i] - b[i];
        prod[i] = a[i] * b[i];
        div[i] = a[i] / b[i];
    }
}

int main(int argc, char* argv[]) {
    int N = 200000;  // Значение по умолчанию

    // Обработка параметров: приоритет у аргументов командной строки
    if (argc > 1) {
        N = atoi(argv[1]);
    } else {
        char *env_n = getenv("ARRAY_SIZE");
        if (env_n != NULL) {
            N = atoi(env_n);
        }
    }

    if (N < 100000) {
        fprintf(stderr, "N must be >= 100000\n");
        return 1;
    }

    double *a, *b, *sum, *diff, *prod, *div;
    double *d_a, *d_b, *d_sum, *d_diff, *d_prod, *d_div;

    // Выделение памяти на хосте
    a = (double*)malloc(N * sizeof(double));
    b = (double*)malloc(N * sizeof(double));
    sum = (double*)malloc(N * sizeof(double));
    diff = (double*)malloc(N * sizeof(double));
    prod = (double*)malloc(N * sizeof(double));
    div = (double*)malloc(N * sizeof(double));

    // Инициализация массивов
    for(int i = 0; i < N; i++) {
        a[i] = i + 1.0;
        b[i] = (i + 1.0) * 2.0;
    }

    // Выделение памяти на устройстве
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_sum, N * sizeof(double));
    hipMalloc(&d_diff, N * sizeof(double));
    hipMalloc(&d_prod, N * sizeof(double));
    hipMalloc(&d_div, N * sizeof(double));

    // Копирование данных на устройство
    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    // Создание событий для замера времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Прогревочный запуск
    array_ops_sequential<<<1, 1>>>(d_a, d_b, d_sum, d_diff, d_prod, d_div, N);
    hipDeviceSynchronize();

    // Замер времени выполнения
    hipEventRecord(start);
    array_ops_sequential<<<1, 1>>>(d_a, d_b, d_sum, d_diff, d_prod, d_div, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Проверка ошибок CUDA
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Расчет времени выполнения
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Вывод результатов
    printf("ВРЕМЯ_ВЫПОЛНЕНИЯ: %.6f s\n", milliseconds / 1000.0f);
    printf("ПОСЛЕДОВАТЕЛЬНАЯ_РЕАЛИЗАЦИЯ\n");
    printf("ЭЛЕМЕНТОВ: %d\n", N);

    // Освобождение памяти
    free(a); free(b); free(sum); free(diff); free(prod); free(div);
    hipFree(d_a); hipFree(d_b); hipFree(d_sum);
    hipFree(d_diff); hipFree(d_prod); hipFree(d_div);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
